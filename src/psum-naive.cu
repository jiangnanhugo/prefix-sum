#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "info.h"


#define THREADS 256  /* Number of per-block threads */


int _length;         /* Length of the input array */
int _size;           /* Size of the input array in bytes */
int _blocks;         /* Number of GPU blocks to use */
int *h_input;        /* Host-side input array */
int *h_output;       /* Host-side output array */
int *d_input;        /* Device-side input array */
int *d_output;       /* Device-side output array */


/* Compute the prefix sum for each element in the block */
__global__ void compute_sums(int *input, int *output)
{
        int tid = threadIdx.x;
        int bid = blockIdx.x;
	int idx = (bid * THREADS) + tid;

        /* Initialize the buffers in shared memory */
        extern __shared__ int shmem[];
        int *in = shmem;
        int *out = &shmem[THREADS];
        in[tid] = input[idx];
        out[tid] = input[idx];
        __syncthreads();

        /* Compute the prefix sums */
        int offset;
        for (offset = 1; offset < THREADS; offset *= 2) {
                /* Swap the arrays */
                int *tmp = in;
                in = out;
                out = tmp;
                __syncthreads();

                if (tid - offset < 0)
                        out[tid] = in[tid];
                else    
                        out[tid] = in[tid] + in[tid - offset];
        }

	/* Copy the shared memory output to main memory */
	output[idx] = out[tid];
}

/* 
 * Add the highest prefix sum of block i-1 to each element
 * of block i
 */
__global__ void aggregate_blocks(int *output, int blocks)
{
	if (blocks == 1)
		return;

	int tid = threadIdx.x;
	int i;
	for (i = 1; i < blocks; i++) {
		int idx = (i * THREADS) + tid;
		int prev_block = (i * THREADS) - 1;
		output[idx] += output[prev_block];
		__syncthreads();
	}
}

/* Parse the input file */
__host__ void read_input(char *inputname)
{
        /* Open the input file */
        FILE *inputfile = fopen(inputname, "r");
        if (inputfile == NULL) {
                fprintf(stderr, "Invalid filename\n");
                free(inputname);
                exit(EXIT_FAILURE);
        }

        /* Read the line count */
        char *line = NULL;
        size_t len = 0;
        ssize_t read = getline(&line, &len, inputfile);
        _length = atoi(line);

	/* Compute the number of blocks to use */
	if (_length <= THREADS)
		_blocks = 1;
	else
		_blocks = _length / THREADS;

	/* Allocate the input/output arrays */
	_size = sizeof(int) * _length;
        h_input = (int *)malloc(_size);
	h_output = (int *)malloc(_size);

        /* Read the input */
        int i = 0;
        while ((read = getline(&line, &len, inputfile)) != -1) {
                int x = atoi(line);
                h_input[i] = x;
                i++;
        }

        /* Copy the input to the GPU */
        hipMalloc((void **) &d_input, _size);
        hipMemcpy(d_input, h_input, _size, 
                   hipMemcpyHostToDevice);

	/* Allocate the output array on the GPU */
	hipMalloc((void **) &d_output, _size);

        free(line);
        fclose(inputfile);
}

/* Print the prefix sums */
__host__ void print_results(int *output, int len)
{
        // int i;
        // for (i = 0; i < len; i++)
        //         printf("%d ", output[i]);
        // printf("\n");

	printf("Final prefix sum: %d\n", output[len - 1]);
}

__host__ int main(int argc, char *argv[])
{
        if (argc < 2) {
                fprintf(stderr, "Must provide a filename\n");
                return -1;
        }
        size_t len = strlen(argv[1]);
        char *inputname = (char *)malloc(len + 1);
        strcpy(inputname, argv[1]);
        read_input(inputname);
        
	/* Compute the prefix sums for each block */
        int shmem_size = sizeof(int) * THREADS * 2;
        compute_sums<<<_blocks, THREADS, shmem_size>>>(d_input, d_output);

	/* Compute the final results */
	aggregate_blocks<<<1, THREADS>>>(d_output, _blocks);
	hipMemcpy(h_output, d_output, _size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	print_results(h_output, _length);

	free(inputname);
	free(h_input);
	free(h_output);
	hipFree(d_input);
	hipFree(d_output);
	return 0;
}
