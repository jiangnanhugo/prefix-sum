#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "info.h"


#define THREADS 256


int length;              /* Length of the input array */
int blocks;              /* Number of GPU blocks to use */
int buffers_size;        /* Size of the input/output buffers in bytes */
int scalars_size;        /* Size of the scalars array in bytes */
int shmem_size;          /* Size of shared memory in bytes */
int *h_input;            /* Host-side input array */
int *h_scalars;          /* Host-side scalars for each block */
int *h_output;           /* Host-side output array */
int *d_input;            /* Device-side input array */
int *d_scalars;          /* Device-side scalars for each block */
int *d_output;           /* Device-side output array */


/* Compute partial prefix sums on the GPU */
__global__ void compute_sums(int *input, int *output, int *scalars, int length)
{
        int tid = threadIdx.x;
        int bid = blockIdx.x + (blockIdx.y * gridDim.x);
        int idx = tid + (bid * blockDim.x);
        if (idx >= length)
                return;

        /* Load input into shared memory */
        extern __shared__ int temp[];
        int *in = temp;
        int *out = &temp[THREADS];
        in[tid] = input[idx];
        out[tid] = input[idx];
        __syncthreads();
        
        /* Compute the partial sums */
        int offset;
        for (offset = 1; offset < THREADS; offset *= 2) {
                /* Swap the buffer pointers */
                int *swap = in;
                in = out;
                out = swap;
                
                if (tid < offset)
                        out[tid] = in[tid];
                else
                        out[tid] = in[tid] + in[tid - offset];
                __syncthreads();
        }
        output[idx] = out[tid];

        /* Copy the highest sum to the array of block sums */
        if (tid == 0)
                scalars[bid] = out[THREADS - 1];
}

/* Add the scalar to each element in the block */
__global__ void add_scalars(int *output, int *scalars, int length)
{
        int tid = threadIdx.x;
        int bid = blockIdx.x + (blockIdx.y * gridDim.x);
        int idx = tid + (bid * blockDim.x);
        if (idx >= length)
                return;

        extern __shared__ int scalar[];
        scalar[0] = scalars[bid - 1];

        if (bid > 0)
                output[idx] += scalar[0];
}

/* Aggregate the scalars on the CPU */
__host__ void compute_scalars(int *scalars, int length)
{
        int i;
        for (i = 1; i < length; i++)
                scalars[i] += scalars[i - 1]; 
}

/* Parse the input file */
__host__ void read_input(char *inputname)
{
        /* Open the input file */
        FILE *inputfile = fopen(inputname, "r");
        if (inputfile == NULL) {
                fprintf(stderr, "Invalid filename\n");
                free(inputname);
                exit(EXIT_FAILURE);
        }

        /* Read the line count */
        char *line = NULL;
        size_t len = 0;
        ssize_t read = getline(&line, &len, inputfile);
        length = atoi(line);

        /* Compute the number of blocks to use */
        if (length <= THREADS)
                blocks = 1;
        else
                blocks = ceil(length / THREADS);
        scalars_size = sizeof(int) * blocks;
        shmem_size = sizeof(int) * THREADS * 2;

        /* Allocate the CPU buffers */
        buffers_size = sizeof(int) * length;
        h_input = (int *)malloc(buffers_size);
        h_output = (int *)malloc(buffers_size);
        h_scalars = (int *)malloc(scalars_size);
        
        /* Read the input */
        int i = 0;
        while ((read = getline(&line, &len, inputfile)) != -1) {
                int x = atoi(line);
                h_input[i] = x;
                i++;
        }

        /* Allocate the GPU buffers */
        hipMalloc((void **) &d_input, buffers_size);
        hipMalloc((void **) &d_output, buffers_size);
        hipMalloc((void **) &d_scalars, scalars_size);
        hipMemcpy(d_input, h_input, buffers_size, hipMemcpyHostToDevice);
        
        free(line);
        fclose(inputfile);
}

__host__ int main(int argc, char *argv[])
{
        if (argc < 2) {
                fprintf(stderr, "Must provide a filename\n");
                return -1;
        }
        size_t len = strlen(argv[1]);
        char *inputname = (char *)malloc(len + 1);
        strcpy(inputname, argv[1]);
        read_input(inputname);
        
        /* Compute the partial sums */
        dim3 grid(ceil(sqrt(blocks)), ceil(sqrt(blocks)));
        dim3 block(THREADS, 1);
        compute_sums<<<grid, block, shmem_size>>>(d_input, d_output,
                                                  d_scalars, length);
        hipMemcpy(h_scalars, d_scalars, scalars_size,
                   hipMemcpyDeviceToHost);

        /* Compute the scalars for each block */
        compute_scalars(h_scalars, blocks);
        hipMemcpy(d_scalars, h_scalars, scalars_size,
                   hipMemcpyHostToDevice);

        /* Add the scalars for each block to the output */
        add_scalars<<<grid, block, sizeof(int)>>>(d_output, d_scalars, length);
        hipMemcpy(h_output, d_output, buffers_size, hipMemcpyDeviceToHost);
        printf("Final prefix sum: %d\n", h_output[length - 1]);
        
        free(inputname);
        free(h_input);
        free(h_output);
        hipFree(d_input);
        hipFree(d_output);
        return 0;
}
